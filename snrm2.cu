#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define n 32

int main()
{
	hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;

    int i;
    float* x;
    float* result;
    
    x = (float*)malloc(n*sizeof(*x));
    
    for (i=0;i<n;i++)
    {
        x[i] = (float)i;
    }
    
    float* d_x;

    hipMalloc((void**)&d_x,n*sizeof(*x));

    stat = hipblasCreate(&handle);
    stat = hipblasSetVector(n,sizeof(*x),x,1,d_x,1);
    
    stat = hipblasSnrm2(handle,n ,d_x, 1,result);
    

    printf("Euclidean norm of x:%f",*result);
    printf("\n");

    hipFree(d_x);
    hipblasDestroy(handle);
    free(x);
    
	return EXIT_SUCCESS;
}

