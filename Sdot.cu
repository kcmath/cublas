#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include "hipblas.h"
#define n 9999999

int main()
{
	hipError_t cudaStat;
	hipblasStatus_t stat;
	hipblasHandle_t handle;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	int j;
	float* x;
	float* y;

	x=(float*)malloc(n*sizeof(*x));
	y=(float*)malloc(n*sizeof(*y));

	for(j=0;j<n;j++)
	{
		x[j]=(float)j;
		y[j]=(float)j;	
	}

	//device
	float* d_x;
	float* d_y;
	float* result;
	float* d_result;

	cudaStat=hipMalloc((void**)&d_x,n*sizeof(*x));
	cudaStat=hipMalloc((void**)&d_y,n*sizeof(*y));
	cudaStat=hipMalloc((void**)&d_result,sizeof(*d_result));
	stat = hipblasCreate(&handle);
	stat = hipblasSetPointerMode(handle,HIPBLAS_POINTER_MODE_DEVICE);
	stat = hipblasSetVector(n,sizeof(*x),x,1,d_x,1);
	stat = hipblasSetVector(n,sizeof(*y),y,1,d_y,1);

	hipEventRecord(start);
	stat = hipblasSdot(handle,n,d_x,1,d_y,1,d_result);
	hipEventRecord(stop);
	hipblasGetVector(1,sizeof(float),d_result,1,result,1);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("dot product x.y\n");
	printf("%f\n",*result);
	printf("GPU time: %f sec", milliseconds);

	hipFree(d_x);
	hipFree(d_y);
	free(x);
	free(y);

	return 0;
}

