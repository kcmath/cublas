#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define m 6
#define n 5

int main()
{
	hipError_t cudaStat;
	hipblasStatus_t stat;
	hipblasHandle_t handle;

	int i, j;
	float* a;
	float* x;
	float* y;
	a = (float*)malloc(m*n*sizeof(float));
	x = (float*)malloc(n*sizeof(float));
	y = (float*)malloc(m*sizeof(float));

	int ind = 11;
	for (j = 0; j < n; j++) {
		for (i = 0; i < m; i++) {
			a [j*m+i]=( float ) ind ++;
		}
	}

	printf("a:\n");
	for (i = 0; i < m; i++) {
		for (j = 0; j < n; j++) {
			printf("%4.0f", a [j*m+i]);
		}
		printf("\n");
	}

	for (i = 0; i < n; i++) x[i] = 1.0f;
	for (i = 0; i < m; i++) y[i] = 0.0f;

	float* d_a;
	float* d_x;
	float* d_y;

	cudaStat = hipMalloc((void**)&d_a,m*n*sizeof(*a));
	cudaStat = hipMalloc((void**)&d_x,n*sizeof(*x));
	cudaStat = hipMalloc((void**)&d_y,m*sizeof(*y));

	stat = hipblasCreate(&handle);
	stat = hipblasSetMatrix(m,n,sizeof(*a), a, m, d_a, m);
	stat = hipblasSetVector(n,sizeof(*x),x,1,d_x,1);
	stat = hipblasSetVector(m,sizeof(*y),y,1,d_y,1);

	float al = 1.0f;
	float bet =0.0f;

	stat = hipblasSgemv(handle, HIPBLAS_OP_N,m,n,&al,d_a,m,d_x,1,&bet,d_y,1);
	stat = hipblasGetVector(m,sizeof(*y),d_y,1,y,1);
	printf ( "Sgemv ::\n" );
	for ( j =0; j < m ; j ++){
		printf ( "%5.0f\n",y[j]);
	}

	hipFree (d_a);
	hipFree (d_x);
	hipFree (d_y);
	hipblasDestroy(handle);
	free(a);
	free(x);
	free(y);

	return EXIT_SUCCESS;
}
