#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define m 6
#define n 4
#define k 5
int main ()
{
	hipError_t cudaStat;
	hipblasStatus_t stat;
	hipblasHandle_t handle;

	int i,j;
	float* a;
	float* b;
	float* c;

	a = (float*)malloc(m*k*sizeof(float));
	b = (float*)malloc(k*n*sizeof(float));
	c = (float*)malloc(m*n*sizeof(float));

	int ind = 11;
	for (j = 0; j < k; j++) {
		for (i = 0; i < m; i++) {
			a [j*m+i] = (float)ind++;
		}
	}

	printf("a:\n");
	for (i = 0; i < m; i++) {
		for (j = 0; j < k; j++) {
			printf("%4.0f", a[j*m+i]);
		}
		printf("\n");
	}

	ind = 11;
	for (j = 0; j < n; j++) {
		for (i = 0; i < k; i++) {
			b [j*m+i] = (float)ind++;
		}
	}

	printf("b:\n");
	for (i = 0; i < k; i++) {
		for (j = 0; j < n; j++) {
			printf("%4.0f", b[j*m+i]);
		}
		printf("\n");
	}

	ind = 11;
	for (j = 0; j < n; j++) {
		for (i = 0; i < m; i++) {
			c [j*m+i] = (float)ind++;
		}
	}

	printf("c:\n");
	for (i = 0; i < m; i++) {
		for (j = 0; j < n; j++) {
			printf("%4.0f", c[j*m+i]);
		}
		printf("\n");
	}

	float* d_a;
	float* d_b;
	float* d_c;


cudaStat = hipMalloc((void**)&d_a,m*k*sizeof(*a));
cudaStat = hipMalloc((void**)&d_b,k*n*sizeof(*b));
cudaStat = hipMalloc((void**)&d_c,m*n*sizeof(*c));

stat = hipblasCreate(&handle);
stat = hipblasSetMatrix(m,k,sizeof(*a),a,m,d_a,m);
stat = hipblasSetMatrix(k,n,sizeof(*b),b,k,d_b,k);
stat = hipblasSetMatrix(m,n,sizeof(*c),c,m,d_c,m);

float al=1.0f;
float bet=1.0f;

stat = hipblasSgemm (handle,HIPBLAS_OP_N,HIPBLAS_OP_N,m,n,k,&al,d_a,m,d_b,k,&bet,d_c,m);

stat = hipblasGetMatrix(m,n,sizeof(*c),d_c,m,c,m);

printf("c:\n");
for (i = 0; i < m; i++) {
	for (j = 0; j < n; j++) {
		printf("%7.0f", c[j*m+i]);
	}
	printf("\n");
}
	return EXIT_SUCCESS;
}
