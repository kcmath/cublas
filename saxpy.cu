#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define n 32

int main()
{
	hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;

    int i;
    float* x;
    float* y;
    
    x = (float*)malloc(n*sizeof(*x));
    y = (float*)malloc(n*sizeof(*y));
    
    for (i=0;i<n;i++)
    {
        x[i] = (float)i;
        y[i] = (float)i;
    }
    
    float* d_x;
    float* d_y;
    float alpha=1.0;

    hipMalloc((void**)&d_x,n*sizeof(*x));
    hipMalloc((void**)&d_y,n*sizeof(*y));

    stat = hipblasCreate(&handle);
    stat = hipblasSetVector(n,sizeof(*x),x,1,d_x,1);
    stat = hipblasSetVector(n,sizeof(*y),y,1,d_y,1);
    
    stat = hipblasSaxpy(handle,n,&alpha,d_x,1,d_y,1);
    
    stat = hipblasGetVector(n,sizeof(float),d_y,1,y,1);
    printf("y = alpha*x + y\n");
    for(i=0;i<n;i++)
    {
        printf("%f\n",y[i]);
    }

    hipFree(d_x);
    hipFree(d_y);
    hipblasDestroy(handle);
    free(x);
    free(y);
    
	return EXIT_SUCCESS;
}

