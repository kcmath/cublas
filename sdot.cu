#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define n 32

int main()
{
	hipError_t cudaStat;
	hipblasStatus_t stat;
	hipblasHandle_t handle;
	
	int j;
	float* x;
	float* y;

	x=(float*)malloc(n*sizeof(*x));
	y=(float*)malloc(n*sizeof(*y));

	for(j=0;j<n;j++)
	{
		x[j]=(float)j;
		y[j]=(float)j;	
	}

	float* d_x;
	float* d_y;
	float* result;

	cudaStat=hipMalloc((void**)&d_x,n*sizeof(*x));
	cudaStat=hipMalloc((void**)&d_y,n*sizeof(*y));

	stat = hipblasCreate(&handle);
	stat = hipblasSetVector(n,sizeof(*x),x,1,d_x,1);
	stat = hipblasSetVector(n,sizeof(*y),y,1,d_y,1);

	stat = hipblasSdot(handle,n,d_x,1,d_y,1,result);

	printf("dot product x.y\n");
	printf("%f\n",*result);

	hipFree(d_x);
	hipFree(d_y);

	free(x);
	free(y);

	return 0;
}

